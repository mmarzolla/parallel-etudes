/****************************************************************************
 *
 * cuda-dot-shared.cu - Dot product with CUDA using __shared__ memory
 *
 * Copyright (C) 2017--2021 by Moreno Marzolla <moreno.marzolla(at)unibo.it>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *    http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 * ---------------------------------------------------------------------------
 *
 * Compile with:
 * nvcc cuda-dot-shared.cu -o cuda-dot-shared -lm
 *
 * Run with:
 * ./cuda-dot-shared [len]
 *
 * Example:
 * ./cuda-dot-shared
 *
 ****************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define BLKDIM 1024

__global__ void dot( double *x, double *y, int n, double *result )
{
    __shared__ double sums[BLKDIM];
    double local_sum = 0.0;

    const int tid = threadIdx.x;
    int i;

    for (i = tid; i < n; i += blockDim.x) {
        local_sum += x[i] * y[i];
    }
    sums[tid] = local_sum;
    __syncthreads(); /* Wait for all threads to write to the shared array */
    /* Thread 0 makes the final reduction */
    if ( 0 == tid ) {
        double sum = 0.0;
        for (i=0; i<blockDim.x; i++) {
            sum += sums[i];
        }
        *result = sum;
    }
}

void vec_init( double *x, double *y, int n )
{
    int i;
    const double tx[] = {1.0/64.0, 1.0/128.0, 1.0/256.0};
    const double ty[] = {1.0, 2.0, 4.0};
    const size_t arrlen = sizeof(tx)/sizeof(tx[0]);

    for (i=0; i<n; i++) {
        x[i] = tx[i % arrlen];
        y[i] = ty[i % arrlen];
    }
}

int main( int argc, char* argv[] )
{
    double *x, *y, result;              /* host copies of x, y, result */
    double *d_x, *d_y, *d_result;       /* device copies of x, y, result */
    int n = 1024*1024;
    const int max_len = 64 * n;

    if ( argc > 2 ) {
        fprintf(stderr, "Usage: %s [len]\n", argv[0]);
        return EXIT_FAILURE;
    }

    if ( argc > 1 ) {
        n = atoi(argv[1]);
    }

    if ( n > max_len ) {
        fprintf(stderr, "FATAL: the maximum length is %d\n", max_len);
        return EXIT_FAILURE;
    }

    const size_t size = n * sizeof(*x);

    /* Allocate space for device copies of x, y, result */
    hipMalloc((void **)&d_x, size);
    hipMalloc((void **)&d_y, size);
    hipMalloc((void **)&d_result, sizeof(*d_result));

    /* Allocate space for host copies of x, y */
    x = (double*)malloc(size);
    y = (double*)malloc(size);
    vec_init(x, y, n);

    /* Copy inputs to device */
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    /* Launch dot() kernel on GPU */
    printf("Computing the dot product of %d elements... ", n);
    dot<<<1, BLKDIM>>>(d_x, d_y, n, d_result);

    /* Copy result back to host */
    hipMemcpy(&result, d_result, sizeof(*d_result), hipMemcpyDeviceToHost);

    printf("result=%f\n", result);
    const double expected = ((double)n)/64;

    /* Check result */
    if ( fabs(result - expected) < 1e-5 ) {
        printf("Check OK\n");
    } else {
        printf("Check FAILED: got %f, expected %f\n", result, expected);
    }

    /* Cleanup */
    free(x); free(y);
    hipFree(d_x); hipFree(d_y); hipFree(d_result);
    return EXIT_SUCCESS;
}
