#include "hip/hip_runtime.h"
/******************************************************************************
 *
 * cuda-coupled-oscillators.c - One-dimensional coupled oscillators
 *
 * Copyright (C) 2017--2021, 2023 by Moreno Marzolla <moreno.marzolla(at)unibo.it>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *    http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 ******************************************************************************/

/***
% HPC - One-dimensional coupled oscillators
% Moreno Marzolla <moreno.marzolla@unibo.it>
% Ultimo aggiornamento: 2023-03-27

![](coupled_metronomes.jpg)

Let us consider $n$ points of mass $m$ arranged along a straight line
at coordinates $x_0, x_1, \ldots, x_{n-1}$. Adjacent masses are
connected by a spring with elastic constant $k$ and rest length
$L$. The first and last points (those in position $x_0$ and $x_{n-1}$
occupy a fixed position and cannot move.

![Figur3 1: Coupled oscillators](cuda-coupled-oscillators.svg)

Initially, one of the springs is displaced so that a wave of
oscillations is triggered; due to the lack of friction, such
oscillations will go on indefinitely. Using Newton's second law of
motion $F = ma$ and Hooke's law which states that a spring with
elastic parameter $k$ that is compressed by $\Delta x$ exerts a force
$k \Delta x$, we develop a program that, given the initial positions
and velocities, computes the positions and speeds of all masses at any
time $t > 0$. The program is based on an iterative algorithm that,
from positions and speeds of the masses at time $t$, determine the new
positions and velocities at time $t + \Delta t$. In particular, the
function

```C
step(double *x, double *v, double *xnext, double *vnext, int n)
```

computes the new position `xnext[i]` and velocity `vnext[i]` of mass
$i$ at time $t + \Delta t$, $0 \le i < n$, given the current position
`x[i]` and velocity `v[i]` at time $t$.

1. For each $i = 1, \ldots, n-2$, the force $F_i$ acting on mass $i$
   is $F_i := k \times (x_{i-1} -2x_i + x_{i+1})$; note that the force
   does not depend on the length $L$ of the spring at rest. Masses 0
   and $n-1$ are stationary, therefore the forces acting on them are
   not computed.

2. For each $i = 1, \ldots, n-2$ the new velocity $v'_i$ of mass $i$
   at time $t + \Delta t$ is $v'_i := v_i + (F_i / m) \Delta
   t$. Again, masses 0 and $n-1$ are statioary, therefore their
   velocities are always zero.

3. For each $i = 1, \ldots, n-2$ the new position $x'_i$ of mass $i$
   at time $t + \Delta t$ is $x'_i := x_i + v'_i \Delta t$. Masses 0
   and $n-1$ are stationary, therefore their positions at time $t +
   \Delta t$ are the same as those at time $t$: $x'_0 := x_0$,
   $x'_{n-1} := x_{n-1}$.

The file [cuda-coupled-oscillators.cu](cuda-coupled-oscillators.cu)
contains a serial program that computes the evolution of $n$ coupled
oscillators. The program produces a two-dimensional image
`coupled-oscillators.ppm` where each line shows the potential energies
of the springs at any time (Figure 2).

![Figura 2: energia potenziale delle molle](coupled-oscillators.png)

Your task is to parallelize function `step()` by defining additional
CUDA kernel(s).

To compile:

        nvcc cuda-coupled-oscillators.cu -o cuda-coupled-oscillators -lm

To execute:

        ./cuda-coupled-oscillators [N]

Example:

        ./cuda-coupled-oscillators 1024

## Files

- [cuda-coupled-oscillators.cu](cuda-coupled-oscillators.cu)
- [hpc.h](hpc.h)

 ***/
#include "hpc.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <assert.h>

/* Number of initial steps to skip, before starting to take pictures */
#define TRANSIENT 50000
/* Number of steps to record in the picture */
#define NSTEPS 800

#ifndef SERIAL
#define BLKDIM 1024
#endif

/* Some physical constants; note that these are defined as symbolic
   values rather than constants, since they must be visible inside a
   kernel functions (and normal constants are not, unless they are
   stored in constant memory on the device) */
/* Integration time step */
#define dt 0.02f
/* spring constant (large k = stiff spring, small k = soft spring) */
#define k 0.2f
/* mass */
#define m 1.0f
/* Length of each spring at rest */
#define L 1.0f

/* Initial conditions: all masses are evenly placed so that the
   springs are at rest; some of the masses are displaced to start the
   movement. */
void init( float *x, float *v, int n )
{
    int i;
    for (i=0; i<n; i++) {
        x[i] = i*L;
        v[i] = 0.0;
    }
    /* displace some of the masses */
    x[n/3  ] -= 0.5*L;
    x[n/2  ] += 0.7*L;
    x[2*n/3] -= 0.7*L;
}

/**
 * Perform one simulation step: starting from the current positions
 * `x[]` and velocities `v[]` of the masses, compute the next
 * positions `xnext[]` and velocities `vnext[]`.
 */
#ifdef SERIAL
void step( float *x, float *v, float *xnext, float *vnext, int n )
{
    int i;
    for (i=0; i<n; i++) {
        if ( i > 0 && i < n - 1 ) {
            /* Compute the net force acting on mass i */
            const float F = k*(x[i-1] - 2*x[i] + x[i+1]);
            const float a = F/m;
            /* Compute the next position and velocity of mass i */
            vnext[i] = v[i] + a*dt;
            xnext[i] = x[i] + vnext[i]*dt;
        } else {
            xnext[i] = x[i];
            vnext[i] = 0.0;
        }
    }
}
#else
__global__ void step( float *x, float *v, float *xnext, float *vnext, int n )
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i >= n )
        return;

    if ( i > 0 && i < n - 1 ) {
        /* Compute the net force acting on mass i */
        const float F = k*(x[i-1] - 2*x[i] + x[i+1]);
        const float a = F/m;
        /* Compute the next position and velocity of mass i */
        vnext[i] = v[i] + a*dt;
        xnext[i] = x[i] + vnext[i]*dt;
    } else {
        /* First and last values of x and v are just copied to the new arrays; */
        xnext[i] = x[i];
        vnext[i] = 0.0;
    }
}
#endif

/**
 * Compute x*x
 */
float squared(float x)
{
    return x*x;
}

/**
 * Compute the maximum energy among all springs.
 */
float maxenergy(const float *x, int n)
{
    int i;
    float maxenergy = -INFINITY;
    for (i=1; i<n; i++) {
        maxenergy = fmaxf(0.5*k*squared(x[i]-x[i-1]-L), maxenergy);
    }
    return maxenergy;
}

void dumpenergy(FILE *fout, const float *x, int n, float maxen)
{
    int i;
    /* Dump spring energies (light color = high energy) */
    maxen = maxenergy(x, n);
    for (i=1; i<n; i++) {
        const float displ = x[i] - x[i-1] - L;
        const float energy = 0.5*k*squared(displ);
        const float v = fminf(energy/maxen, 1.0);
        fprintf(fout, "%c%c%c", 0, (int)(255*v*(displ<0)), (int)(255*v*(displ>0)));
    }
}

int main( int argc, char *argv[] )
{
    int s, cur = 0, next;
    float maxen;
    int N = 1024;
    const char* fname = "coupled-oscillators.ppm";
#ifdef SERIAL
    float *x[2], *v[2];
#else
    float *x, *v;
    float *d_x[2], *d_v[2];
    const int NBLOCKS = (N + BLKDIM-1)/BLKDIM;
#endif

    if (argc > 1) {
        fprintf(stderr, "Usage: %s [N]\n", argv[0]);
        return EXIT_FAILURE;
    }

    if (2 == argc) {
        N = atoi(argv[1]);
    }

    const size_t size = N * sizeof(float);

    FILE *fout = fopen(fname, "w");
    if (NULL == fout) {
        printf("Cannot open %s for writing\n", fname);
        return EXIT_FAILURE;
    }

    /* Write the header of the output file */
    fprintf(fout, "P6\n");
    fprintf(fout, "%d %d\n", N-1, NSTEPS);
    fprintf(fout, "255\n");

#ifdef SERIAL
    x[0] = (float*)malloc(size); assert(x[0]);
    x[1] = (float*)malloc(size); assert(x[1]);
    v[0] = (float*)malloc(size); assert(v[0]);
    v[1] = (float*)malloc(size); assert(v[1]);
#else
    /* Allocate host copies of x and v */
    x = (float*)malloc(size); assert(x);
    v = (float*)malloc(size); assert(v);

    /* Allocate device copies of x and v */
    cudaSafeCall( hipMalloc((void**)&d_x[0], size) );
    cudaSafeCall( hipMalloc((void**)&d_x[1], size) );
    cudaSafeCall( hipMalloc((void**)&d_v[0], size) );
    cudaSafeCall( hipMalloc((void**)&d_v[1], size) );
#endif

    /* Initialize the simulation */
#ifdef SERIAL
    init(x[cur], v[cur], N);
#else
    init(x, v, N);

    /* Copy data to device */
    cudaSafeCall( hipMemcpy(d_x[cur], x, size, hipMemcpyHostToDevice) );
    cudaSafeCall( hipMemcpy(d_v[cur], v, size, hipMemcpyHostToDevice) );
#endif

    /* Write NSTEPS rows in the output image */
    for (s=0; s<TRANSIENT + NSTEPS; s++) {
        next = 1 - cur;
#ifdef SERIAL
        step(x[cur], v[cur], x[next], v[next], N);
#else
        step<<<NBLOCKS, BLKDIM>>>(d_x[cur], d_v[cur], d_x[next], d_v[next], N);
        cudaCheckError();
#endif
        if (s >= TRANSIENT) {
#ifdef SERIAL
            if (s == TRANSIENT) {
                maxen = maxenergy(x[next], N);
            }
            dumpenergy(fout, x[next], N, maxen);
#else
            cudaSafeCall( hipMemcpy(x, d_x[next], size, hipMemcpyDeviceToHost) );
            if (s == TRANSIENT) {
                maxen = maxenergy(x, N);
            }
            dumpenergy(fout, x, N, maxen);
#endif
        }
        cur = 1 - cur;
    }

#ifdef SERIAL
    free(x[0]);
    free(x[1]);
    free(v[0]);
    free(v[1]);
#else
    free(x);
    free(v);
    hipFree(d_x[0]);
    hipFree(d_x[1]);
    hipFree(d_v[0]);
    hipFree(d_v[1]);
#endif

    fclose(fout);
    return EXIT_SUCCESS;
}
