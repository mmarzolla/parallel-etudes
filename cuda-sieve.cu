#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * cuda-sieve.c - Sieve of Eratosthenes
 *
 * Copyright (C) 2024 Moreno Marzolla
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 ****************************************************************************/

/***
% HPC - Sieve of Eratosthenes
% [Moreno Marzolla](https://www.unibo.it/sitoweb/moreno.marzolla)
% Last updated: 2024-09-02

## Files

- [cuda-sieve.c](cuda-sieve.cu)
- [hpc.h](hpc.h)

***/

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include "hpc.h"

#ifdef SERIAL
/* Mark all mutliples of `k` in the set {`from`, ..., `to`-1}; return
   how many numbers have been marked for the first time. `from` does
   not need to be a multiple of `k`, although in this program it
   always is. */
int mark( char *isprime, int k, int from, int to )
{
    int nmarked = 0;
    from = ((from + k - 1)/k)*k; /* start from the lowest multiple of p that is >= from */
    for ( int x=from; x<to; x+=k ) {
        if (isprime[x]) {
            isprime[x] = 0;
            nmarked++;
        }
    }
    return nmarked;
}
#else
#define BLKDIM 1024

/**
 * Mark all multiples of k belonging to the set {from, ... to-1}.
 * from must be a multiple of k. The number of elements that are
 * marked for the first time is atomically subtracted from *nprimes.
 */
__global__ void
mark_kernel( char *isprime,
             int k,
             int from,
             int to,
             int *nprimes )
{
    __shared__ int mark[BLKDIM];
    const int i = from + (threadIdx.x + blockIdx.x * blockDim.x)*k;
    const int li = threadIdx.x;

    mark[li] = 0;
    __syncthreads();

    if (i < to) {
        mark[li] = (isprime[i] == 1);
        isprime[i] = 0;
    }

    __syncthreads();

    int d = blockDim.x;
    while (d > 1) {
        int d2 = (d + 1)/2;
        if (li + d2 < d) mark[li] += mark[li + d2];
        d = d2;
        __syncthreads();
    }
    if (0 == li) {
        atomicSub(nprimes, mark[0]);
    }
}

__global__ void
next_prime_kernel(const char *isprime,
                  int k,
                  int n,
                  int *next_prime)
{
    if (threadIdx.x == 0) {
        k++;
        while (k < n && isprime[k] == 0)
            k++;
        *next_prime = k;
    }
}
#endif

int primes(int n)
{
    char *isprime = (char*)malloc(n+1); assert(isprime != NULL);
    int nprimes = n-2;

    /* Initially, all numbers are considered primes */
    for (int i=0; i<=n; i++)
        isprime[i] = 1;

#ifdef SERIAL
    /* main iteration of the sieve */
    for (int i=2; ((long)i)*i <= (long)n; i++) {
        if (isprime[i]) {
            nprimes -= mark(isprime, i, i*i, n+1);
        }
    }
#else
    char *d_isprime;
    int *d_nprimes, *d_next_prime;

    cudaSafeCall( hipMalloc( (void**)&d_isprime, n+1) );
    cudaSafeCall( hipMemcpy( d_isprime, isprime, n+1, hipMemcpyHostToDevice) );

    cudaSafeCall( hipMalloc( (void**)&d_nprimes, sizeof(*d_nprimes)) );
    cudaSafeCall( hipMemcpy( d_nprimes, &nprimes, sizeof(nprimes), hipMemcpyHostToDevice) );

    cudaSafeCall( hipMalloc( (void**)&d_next_prime, sizeof(*d_next_prime)) );

    const dim3 BLOCK(BLKDIM);
    /* main iteration of the sieve */
    int k = 2;
    while (((long)k)*k <= (long)n) {
        const int from = k*k;
        const int to = n;
        const int nelem = (to - from + k-1)/k;
        const dim3 GRID((nelem + BLKDIM - 1)/BLKDIM);
        mark_kernel<<<GRID, BLOCK>>>(d_isprime, k, from, to, d_nprimes); cudaCheckError();
        next_prime_kernel<<<1, 1>>>(d_isprime, k, n, d_next_prime); cudaCheckError();
        const int oldk = k;
        cudaSafeCall( hipMemcpy(&k, d_next_prime, sizeof(k), hipMemcpyDeviceToHost) );
        assert(k > oldk);
    }
    cudaSafeCall( hipMemcpy(&nprimes, d_nprimes, sizeof(nprimes), hipMemcpyDeviceToHost) );
    cudaSafeCall( hipFree(d_nprimes) );
    cudaSafeCall( hipFree(d_isprime) );
#endif
    free(isprime);
    return nprimes;
}

int main( int argc, char *argv[] )
{
    int n = 1000000;

    if ( argc > 2 ) {
        fprintf(stderr, "Usage: %s [n]\n", argv[0]);
        return EXIT_FAILURE;
    }

    if ( argc == 2 ) {
        n = atol(argv[1]);
    }

    const double tstart = hpc_gettime();
    const int nprimes = primes(n);
    const double elapsed = hpc_gettime() - tstart;

    printf("There are %d primes in {2, ..., %d}\n", nprimes, n);

    printf("Execution time: %f\n", elapsed);

    return EXIT_SUCCESS;
}
