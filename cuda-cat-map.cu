#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * cuda-cat-map.cu - Arnold's cat map
 *
 * Copyright (C) 2016--2025 Moreno Marzolla
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 ****************************************************************************/

/***
% HPC - Arnold's cat map
% [Moreno Marzolla](https://www.unibo.it/sitoweb/moreno.marzolla)
% Last updated: 2025-10-09

![](cat-map.png)

[Arnold's cat map](https://en.wikipedia.org/wiki/Arnold%27s_cat_map)
is a continuous chaotic function that has been studied in the '60s by
the Russian mathematician [Vladimir Igorevich
Arnold](https://en.wikipedia.org/wiki/Vladimir_Arnold). In its
discrete version, the function can be understood as a transformation
of a bitmapped image $P$ of size $N \times N$ into a new image $P'$ of
the same size. For each $0 \leq x, y < N$, the pixel of coordinates
$(x,y)$ in $P$ is mapped into a new position $C(x, y) = (x', y')$ in
$P'$ where

$$
x' = (2x + y) \bmod N, \qquad y' = (x + y) \bmod N
$$

("mod" is the integer remainder operator, i.e., operator `%` of the C
language). We assume that $(0, 0)$ is top left and $(N-1, N-1)$ is
bottom right, so that the bitmap can be encoded as a regular
two-dimensional C matrix.

The transformation performs a linear "stretching" of the image, which
is then broken down into triangles that are rearranged as shown in
Figure 1.

![Figure 1: Arnold's cat map](cat-map.svg)

Arnold's cat map has some interesting properties. Let $C^k(x, y)$ be
the result of iterating function $C()$ $k$ times, i.e.:

$$
C^k(x, y) = \begin{cases}
(x, y) & \mbox{if $k=0$}\\
C(C^{k-1}(x,y)) & \mbox{if $k>0$}
\end{cases}
$$

Therefore, $C^2(x,y) = C(C(x,y))$, $C^3(x,y) = C(C(C(x,y)))$, and so
on.

If we apply $C$ to an image, we get a severely distorted version of
the input. If we apply $C$ on the resulting image, we get an even more
distorted image. As we keep applying $C$, the original image is no
longer discernible. However, after a certain number of iterations that
depends on $N$ and has been proved to never exceed $3N$, we get back
the original image! (Figure 2).

![Figure 2: Some iterations of the cat map](cat-map-demo.png)

The _minimum recurrence time_ for an image is the minimum positive
integer $k \geq 1$ such that $C^k(x, y) = (x, y)$ for all $(x, y)$. In
simple terms, the minimum recurrence time is the minimum number of
iterations of the cat map that produce the starting image.

For example, the minimum recurrence time for
[cat1368.pgm](cat1368.pgm) of size $1368 \times 1368$ is $36$. As said
before, the minimum recurrence time depends on the image size $N$.
Unfortunately, no closed formula is known to compute the minimum
recurrence time as a function of $N$, although there are results and
bounds that apply to specific cases.

You are provided with a serial program that computes the $k$-th
iterate of Arnold's cat map on a square image. The program reads the
input from standard input in
[PGM](https://en.wikipedia.org/wiki/Netpbm) (_Portable GrayMap_)
format. The results is printed to standard output in PGM format. For
example:

        ./cuda-cat-map 100 < cat1368.pgm > cat1368-100.pgm

applies the cat map $k=100$ times on `cat1368.phm` and saves the
result to `cat1368-100.pgm`.

To display a PGM image you might need to convert it to a different
format, e.g., JPEG. Under Linux you can use `convert` from the
[ImageMagick](https://imagemagick.org/) package:

        convert cat1368-100.pgm cat1368-100.jpeg

To make use of CUDA parallelism, define a 2D grid of 2D blocks that
covers the input image. The block size is $\mathit{BLKDIM} \times
\mathit{BLKDIM}$, with `BLKDIM = 32`, and the grid size is:

$$
(N + \mathit{BLKDIM} – 1) / \mathit{BLKDIM} \times (N + \mathit{BLKDIM} – 1) / \mathit{BLKDIM}
$$

Each thread applies a single iteration of the cat map and copies one
pixel from the input image to the correct position of the output
image.  The kernel has the following signature:

```C
__global__ void cat_map_iter( unsigned char *cur, unsigned char *next, int N )
```

where $N$ is the height/width of the image. The program must work
correctly even if $N$ is not an integer multiple of _BLKDIM_. Each
thread is mapped to the coordinates $(x, y)$ of a pixel using the
usual formulas:

```C
        const int x = threadIdx.x + blockIdx.x * blockDim.x;
        const int y = threadIdx.y + blockIdx.y * blockDim.y;
```

Therefore, to compute the $k$-th iteration of the cat map we need to
execute the kernel $k$ times.

A better approach is to define a kernel

```C
__global__ void cat_map_iter_k( unsigned char *cur, unsigned char *next, int N, int k )
```

that applies $k$ iterations of the cat map to the current image.  This
kernel needs to be executed only once, and this saves some significant
overhead associated to kernel calls. The new kernel can be
defined as follows:

```C
const int x = ...;
const int y = ...;
int xcur = x, ycur = y, xnext, ynext;

if ( x < N && y < N ) {
	while (k--) {
		xnext = (2*xcur + ycur) % N;
		ynext = (xcur + ycur) % N;
		xcur = xnext;
		ycur = ynext;
	}
	\/\* copy the pixel (x, y) from the current image to
	the position (xnext, ynext) of the new image \*\/
}
```

I suggest to implement both solutions (the one where the kernel is
executed $k$ times, and the one where the kernel is executed only
once) and measure the execution times to see the difference.

To compile:

        nvcc cuda-cat-map.cu -o cuda-cat-map

To execute:

        ./cuda-cat-map k < input_file > output_file

Example:

        ./cuda-cat-map 100 < cat1368.pgm > cat1368.100.pgm

## Files

- [cuda-cat-map.cu](cuda-cat-map.cu)
- [hpc.h](hpc.h)
- [cat1368.pgm](cat1368.pgm) (the minimum recurrence time of this image is 36)

***/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>

#include "hpc.h"
#include "pgmutils.h"

#ifndef SERIAL
#define BLKDIM 32
#endif

#ifndef SERIAL
/**
 * Compute one iteration of the cat map using the GPU
 */
__global__ void cat_map_iter( unsigned char *cur, unsigned char *next, int w, int h )
{
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;

    if ( x < w && y < h ) {
        const int xnext = (2*x+y) % w;
        const int ynext = (x + y) % h;
        next[xnext + ynext*w] = cur[x+y*w];
    }
}

/**
 * Compute `k` iterations of the cat map using the GPU
 */
__global__ void cat_map_iter_k( unsigned char *cur, unsigned char *next, int N, int k )
{
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;

    if ( x < N && y < N ) {
        int xcur = x, ycur = y, xnext, ynext;
        while (k--) {
            xnext = (2*xcur+ycur) % N;
            ynext = (xcur + ycur) % N;
            xcur = xnext;
            ycur = ynext;
        }
        next[xnext + ynext*N] = cur[x+y*N];
    }
}
#endif

/**
 * Compute the `k`-th iterate of the cat map for image `img`. The
 * width and height of the input image must be equal. This function
 * replaces the bitmap of `img` with the one resulting after ierating
 * `k` times the cat map. You need to allocate a temporary image, with
 * the same size of the original one, so that you read the pixel from
 * the "old" image and copy them to the "new" image (this is similar
 * to a stencil computation, as was discussed in class). After
 * applying the cat map to all pixel of the "old" image the role of
 * the two images is exchanged: the "new" image becomes the "old" one,
 * and vice-versa. The temporary image must be deallocated upon exit.
 */
void cat_map( PGM_image* img, int k )
{
    const int N = img->width;
    const size_t size = N * N * sizeof(img->bmap[0]);

#ifdef SERIAL
    /* [TODO] Modify the body of this function to allocate device memory,
       do the appropriate data transfer, and launch a kernel */
    unsigned char *cur = img->bmap;
    unsigned char *next = (unsigned char*)malloc( size );

    assert(next != NULL);
    for (int i=0; i<k; i++) {
        for (int y=0; y<N; y++) {
            for (int x=0; x<N; x++) {
                int xnext = (2*x+y) % N;
                int ynext = (x + y) % N;
                next[xnext + ynext*N] = cur[x+y*N];
            }
        }
        /* Swap old and new */
        unsigned char *tmp = cur;
        cur = next;
        next = tmp;
    }
    img->bmap = cur;
    free(next);
#else
    dim3 block(BLKDIM, BLKDIM);
    dim3 grid((N + BLKDIM-1)/BLKDIM, (N + BLKDIM-1)/BLKDIM);

    unsigned char *d_cur, *d_next;

    assert( img->width == img->height );

    /* Allocate bitmaps on the device */
    hipMalloc((void**)&d_cur, size);
    hipMalloc((void**)&d_next, size);

    /* Copy input image to device */
    hipMemcpy(d_cur, img->bmap, size, hipMemcpyHostToDevice);

#if 0
    /* This version performs k kernel calls */
    while( k-- ) {
        cat_map_iter<<<grid,block>>>(d_cur, d_next, N);
        /* swap cur and next */
        unsigned char *d_tmp = d_cur;
        d_cur = d_next;
        d_next = d_tmp;
    }
    hipMemcpy(img->bmap, d_cur, size, hipMemcpyDeviceToHost);
#else
    /* This version performs one kernel call */
    cat_map_iter_k<<<grid,block>>>(d_cur, d_next, N, k);
    hipMemcpy(img->bmap, d_next, size, hipMemcpyDeviceToHost);
#endif

    /* Free memory on device */
    hipFree(d_cur); hipFree(d_next);
#endif
}

int main( int argc, char* argv[] )
{
    PGM_image img;
    int niter;

    if ( argc != 2 ) {
        fprintf(stderr, "Usage: %s niter < input_image > output_image\n", argv[0]);
        return EXIT_FAILURE;
    }
    niter = atoi(argv[1]);
    read_pgm(stdin, &img);
    if ( img.width != img.height ) {
        fprintf(stderr, "FATAL: width (%d) and height (%d) of the input image must be equal\n", img.width, img.height);
        return EXIT_FAILURE;
    }
    const double tstart = hpc_gettime();
    cat_map(&img, niter);
    const double elapsed = hpc_gettime() - tstart;
    fprintf(stderr, "    Iterations: %d\n", niter);
    fprintf(stderr, "  Width,Height: %d,%d\n", img.width, img.height);
    fprintf(stderr, "      Mops/sec: %.4f\n", 1.0e-6 * img.width * img.height * niter / elapsed);
    fprintf(stderr, "Execution time  %.3f\n", elapsed);

    write_pgm(stdout, &img, "produced by cuda-cat-map.cu");
    free_pgm(&img);
    return EXIT_SUCCESS;
}
