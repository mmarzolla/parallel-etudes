#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * cuda-edge-detect.cu - Edge detection on grayscale images
 *
 * Copyright (C) 2024 Moreno Marzolla
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 ****************************************************************************/

/***
% HPC - Edge detection on grayscale images
% [Moreno Marzolla](https://www.unibo.it/sitoweb/moreno.marzolla)
% Last updated: 2024-09-02

![Result of the Sobel operator](edge-detect.png)

The [Sobel operator](https://en.wikipedia.org/wiki/Sobel_operator) is
used to detect the edges on an grayscale image. The idea is to compute
the gradient of color change across each pixel; those pixels for which
the gradient exceeds a user-defined threshold are considered to be
part of an edge. Computation of the gradient involves the application
of a $3 \times 3$ stencil to the input image.

The program reads an input image fro standard input in
[PGM](https://en.wikipedia.org/wiki/Netpbm#PGM_example) (_Portable
Graymap_) format and produces a B/W image to standard output. The user
can specify an optional threshold on the command line.

The goal of this exercise is to parallelize the computation of the
Sobel operator using CUDA; this can be achieved by writing a kernel
that computes the edge at each pixel, and invoke the kernel from the
`edge_detect()` function.

To compile:

        nvcc cuda-edge-detect.cu -o cuda-edge-detect

To execute:

        ./cuda-edge-detect [threshold] < input > output

Example:

        ./cuda-edge-detect < BWstop-sign.pgm > BWstop-sign-edges.pgm

## Files

- [cuda-edge-detect.cu](cuda-edge-detect.cu) [hpc.h](hpc.h)
- [BWstop-sign.pgm](BWstop-sign.pgm)

***/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>

#include "hpc.h"
#include "pgmutils.h"

#ifndef SERIAL
#define BLKDIM 32

__device__ __host__
#endif
int IDX(int i, int j, int width)
{
    return (i*width + j);
}

#ifndef SERIAL
__global__ void
sobel_kernel(const unsigned char *in,
             unsigned char *edges,
             int width, int height,
             int threshold)
{
    const unsigned char WHITE = 255;
    const unsigned char BLACK = 0;
    const int j = threadIdx.x + blockIdx.x * blockDim.x;
    const int i = threadIdx.y + blockIdx.y * blockDim.y;

    if (i >= height || j >= width)
        return;

    if (i==0 || j==0 || i==height-1 || j==width-1)
        edges[IDX(i, j, width)] = WHITE;
    else {
        /* Compute the gradients Gx and Gy along the x and y
           dimensions */
        const int Gx =
            in[IDX(i-1, j-1, width)] - in[IDX(i-1, j+1, width)]
            + 2*in[IDX(i, j-1, width)] - 2*in[IDX(i, j+1, width)]
            + in[IDX(i+1, j-1, width)] - in[IDX(i+1, j+1, width)];
        const int Gy =
            in[IDX(i-1, j-1, width)] + 2*in[IDX(i-1, j, width)] + in[IDX(i-1, j+1, width)]
            - in[IDX(i+1, j-1, width)] - 2*in[IDX(i+1, j, width)] - in[IDX(i+1, j+1, width)];
        const int magnitude = Gx * Gx + Gy * Gy;
        if  (magnitude > threshold*threshold)
            edges[IDX(i, j, width)] = WHITE;
        else
            edges[IDX(i, j, width)] = BLACK;
    }
}
#endif

/**
 * Edge detection using the Sobel operator
 */
void edge_detect( const PGM_image* in, PGM_image* edges, int threshold )
{
    const int width = in->width;
    const int height = in->height;
#ifdef SERIAL
    for (int i = 1; i < height-1; i++) {
        for (int j = 1; j < width-1; j++)  {
            /* Compute the gradients Gx and Gy along the x and y
               dimensions */
            const int Gx =
                in->bmap[IDX(i-1, j-1, width)] - in->bmap[IDX(i-1, j+1, width)]
                + 2*in->bmap[IDX(i, j-1, width)] - 2*in->bmap[IDX(i, j+1, width)]
                + in->bmap[IDX(i+1, j-1, width)] - in->bmap[IDX(i+1, j+1, width)];
            const int Gy =
                in->bmap[IDX(i-1, j-1, width)] + 2*in->bmap[IDX(i-1, j, width)] + in->bmap[IDX(i-1, j+1, width)]
                - in->bmap[IDX(i+1, j-1, width)] - 2*in->bmap[IDX(i+1, j, width)] - in->bmap[IDX(i+1, j+1, width)];
            const int magnitude = Gx * Gx + Gy * Gy;
            if  (magnitude > threshold*threshold)
                edges->bmap[IDX(i, j, width)] = WHITE;
            else
                edges->bmap[IDX(i, j, width)] = BLACK;
        }
    }
#else
    const size_t size = width * height;
    const dim3 block(BLKDIM, BLKDIM);
    const dim3 grid((width + BLKDIM-1)/BLKDIM, (height + BLKDIM-1)/BLKDIM);
    unsigned char *d_in, *d_edges;
    cudaSafeCall( hipMalloc((void**)&d_in, size) );
    cudaSafeCall( hipMalloc((void**)&d_edges, size) );
    cudaSafeCall( hipMemcpy(d_in, in->bmap, size, hipMemcpyHostToDevice) );
    sobel_kernel<<< grid, block >>>(d_in, d_edges, width, height, threshold);
    cudaSafeCall( hipMemcpy(edges->bmap, d_edges, size, hipMemcpyDeviceToHost) );
    cudaSafeCall( hipFree(d_in) );
    cudaSafeCall( hipFree(d_edges) );
#endif
}

int main( int argc, char* argv[] )
{
    PGM_image bmap, out;
    int threshold = 70;

    if ( argc > 2 ) {
        fprintf(stderr, "Usage: %s [threshold] < in.pgm > out.pgm\n", argv[0]);
        return EXIT_FAILURE;
    }
    if ( argc > 1 ) {
        threshold = atoi(argv[1]);
    }
    read_pgm(stdin, &bmap);
    init_pgm(&out, bmap.width, bmap.height, WHITE);
    const double tstart = hpc_gettime();
    edge_detect(&bmap, &out, threshold);
    const double elapsed = hpc_gettime() - tstart;
    fprintf(stderr, "Execution time %f\n", elapsed);
    write_pgm(stdout, &out, "produced by opencl-edge-detect.c");
    free_pgm(&bmap);
    free_pgm(&out);
    return EXIT_SUCCESS;
}
